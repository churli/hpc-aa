#include <hip/hip_runtime.h>

#include "stdio.h"

#define TILE_SIZE 512
#define WARP_SIZE 32

extern "C" void CSRmatvecmult(int* ptr, int* J, float* Val, int N, int nnz, float* x, float *y, bool bVectorized);
extern "C" void ELLmatvecmult(int N, int num_cols_per_row , int * indices, float * data , float * x , float * y);

/**
 * Custom CUDA error check wrapper.
 */
#define checkCUDAError() do {                           \
    hipError_t error = hipGetLastError();               \
    if (error != hipSuccess) {                            \
        printf("(CUDA) %s", hipGetErrorString(error)); \
        printf(" (" __FILE__ ":%d)\n", __LINE__);  \
    }\
} while (0)

/**
 * Cuda kernel for: CSR_s(A)x = y
 */
__global__ void k_csr_mat_vec_mm(int *ptr, int* indices, float *data, int num_rows, float *x, float* y) {
    //TODO: implement the CSR kernel
}

/**
 * Cuda kernel for: CSR_v(A)x = y
 */
__global__ void k_csr2_mat_vec_mm(int *ptr, int* indices, float *data, int num_rows, float *x, float* y) {
    //TODO: implement the vectorized CSR kernel
}

/**
 * Cuda kernel for: ELL(A)x = y
 */
__global__ void k_ell_mat_vec_mm ( int N, int num_cols_per_row , int * indices, float * data , float * x , float * y ) {
    //NYI: ellpack kernel
}

/**
 * Perform: CSR(A)x = y
 */
void CSRmatvecmult(int* ptr, int* J, float* Val, int N, int nnz, float* x, float *y, bool bVectorized) {
    int *ptr_d, *J_d;
    float *Val_d, *x_d, *y_d;

    /************************/
    /* copy to device       */
    /************************/

    hipMalloc((void **) &ptr_d, (N+1) * sizeof(int));
    checkCUDAError();
    hipMemcpy(ptr_d, ptr, (N+1) * sizeof(int), hipMemcpyHostToDevice);
    checkCUDAError();

    hipMalloc((void **) &J_d, nnz * sizeof(int));
    checkCUDAError();
    hipMemcpy(J_d, J, nnz * sizeof(int), hipMemcpyHostToDevice);
    checkCUDAError();

    hipMalloc((void **) &Val_d, nnz * sizeof(float));
    checkCUDAError();
    hipMemcpy(Val_d, Val, nnz * sizeof(float), hipMemcpyHostToDevice);
    checkCUDAError();

    hipMalloc((void **) &x_d, N * sizeof(float));
    checkCUDAError();
    hipMemcpy(x_d, x, N * sizeof(float), hipMemcpyHostToDevice);
    checkCUDAError();

    hipMalloc((void **) &y_d, N * sizeof(float));
    checkCUDAError();
    hipMemcpy(y_d, y, N * sizeof(float) , hipMemcpyHostToDevice);
    checkCUDAError();

    /************************/
    /* start kernel         */
    /************************/

    if (bVectorized) {
        //TODO: define grid and block size correctly
        dim3 grid(0, 0, 0);
        dim3 block(0, 0, 0);

        k_csr2_mat_vec_mm <<< grid, block >>> (ptr_d, J_d, Val_d, N, x_d, y_d);
    } else {
        dim3 grid((N - 1)/TILE_SIZE + 1, 1, 1);
        dim3 block(TILE_SIZE, 1, 1);

        k_csr_mat_vec_mm <<< grid, block >>> (ptr_d, J_d, Val_d, N, x_d, y_d);
    }

    checkCUDAError();

    /************************/
    /* copy back            */
    /************************/

    hipMemcpy(y, y_d, N * sizeof(float), hipMemcpyDeviceToHost);
    checkCUDAError();

    /************************/
    /* free memory          */
    /************************/
    hipFree(ptr_d);
    hipFree(J_d);
    hipFree(Val_d);
    hipFree(x_d);
    hipFree(y_d);
}

/**
 * Perform: ELL(A)x = y
 */
void ELLmatvecmult(int N, int num_cols_per_row , int * indices,
        float * data , float * x , float * y) {
    int *indices_d;
    float *data_d, *x_d, *y_d;

    /************************/
    /* copy to device       */
    /************************/

    hipMalloc((void **) &indices_d, N * num_cols_per_row * sizeof(int));
    checkCUDAError();
    hipMemcpy(indices_d, indices, N * num_cols_per_row * sizeof(int), hipMemcpyHostToDevice);
    checkCUDAError();

    hipMalloc((void **) &data_d, N * num_cols_per_row * sizeof(float));
    checkCUDAError();
    hipMemcpy(data_d, data, N * num_cols_per_row * sizeof(float), hipMemcpyHostToDevice);
    checkCUDAError();

    hipMalloc((void **) &x_d, N * sizeof(float));
    checkCUDAError();
    hipMemcpy(x_d, x, N * sizeof(float), hipMemcpyHostToDevice);
    checkCUDAError();

    hipMalloc((void **) &y_d, N * sizeof(float));
    checkCUDAError();
    hipMemcpy(y_d, y, N * sizeof(float), hipMemcpyHostToDevice);
    checkCUDAError();

    /************************/
    /* start kernel         */
    /************************/

    //NYI: define grid and block size
    //k_ell_mat_vec_mm <<< grid, block >>> (N, num_cols_per_row, indices_d, data_d , x_d, y_d);
    checkCUDAError();

    /************************/
    /* copy back            */
    /************************/

    hipMemcpy(y, y_d, N * sizeof(float), hipMemcpyDeviceToHost);
    checkCUDAError();

    /************************/
    /* free memory          */
    /************************/

    hipFree(indices_d);
    hipFree(data_d);
    hipFree(x_d);
    hipFree(y_d);
}

